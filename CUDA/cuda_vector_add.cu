
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <assert.h>

__global__ void cuda_vector_add(int *a, int *b)
{
	__shared__ int results[64];  // Actually we don't need this, just for illustration
	
	int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	int local_thread_id  = threadIdx.x;
	
	results[local_thread_id] = a[global_thread_id] + b[global_thread_id];
	
	__syncthreads();
	
	a[global_thread_id] = results[local_thread_id];
}

int main(int argc, char **argv)
{
	int n, nBytes;
	n = atoi(argv[1]);
	n = (n + 63) / 64 * 64;
	nBytes = sizeof(int) * n;
	printf("Vector add, length = %d\n", n);
	
	int *h_a, *h_b, *d_a, *d_b;
	// Allocate memory on host
	h_a = (int*) malloc(nBytes);
	h_b = (int*) malloc(nBytes);
	// Allocate memory on device
	hipMalloc((void**) &d_a, nBytes);
	hipMalloc((void**) &d_b, nBytes);
	
	// Init data on host
	for (int i = 0; i < n; i++)
	{
		h_a[i] = 114 + i;
		h_b[i] = 514 - i;
	}
	
	// Copy data to device
	hipMemcpy(d_a, h_a, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, nBytes, hipMemcpyHostToDevice);
	
	// Set kernel arguments and launch kernel
	dim3 block(64);
	dim3 grid(n / block.x);
	cuda_vector_add<<<grid, block>>>(d_a, d_b);
	
	// Generate result on host
	for (int i = 0; i < n; i++)	h_b[i] += h_a[i];
	
	// Copy result from device to host
	hipMemcpy(h_a, d_a, nBytes, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	
	// Check the results
	for (int i = 0; i < n; i++) assert(h_a[i] == h_b[i]);
	printf("Result is correct.\n");
	
	// Free host memory
	free(h_a);
	free(h_b);
	
	// Free device memory
	hipFree(d_a);
	hipFree(d_b);
	
	return 0;
}